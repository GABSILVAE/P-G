﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include <unistd.h>
#include <fcntl.h>
#include <poll.h>

using namespace std;

#define SYSFS_GPIO_DIR "/sys/class/gpio"
#define POLL_TIMEOUT (3 * 1000) /* 3 seconds */
#define MAX_BUF 64

__host__ void writeFile(char *fileName, char line[]) {
	FILE* file;
	file = fopen(fileName, "w");

	fputs(line, file);

	fclose(file);
}

int main(int *argc, char** argv[]) {
	printf("Script for writting a line to a txt file");

	char line[255] = "Helo world from Host";

	writeFile("test.txt", line);

	return 0;
}
