﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

using namespace std;

__host__ void writeFile(char *fileName, char line[]) {
	FILE* file;
	file = fopen(fileName, "w");

	fputs(line, file);

	fclose(file);
}

int main(int *argc, char** argv[]) {
	printf("Script for writting a line to a txt file");

	char line[255] = "Helo world from Host";

	writeFile("test.txt", line);

	return 0;
}