#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdio.h>
#include<hip/hip_runtime.h>

#define threads 10 // numero de hilos por bloque 
__global__void suma(int *vector_1, int *vector_2, int *verctor_suma,int n){

	// kernel multibloque 
	// identificaador global de cada hilo en el bloque 
	int index =trheadIdx.x + blockDim.x*blockIdx;
	
	//truncamiento de los hilos solo trabajan los hilos nesesarios 
	if(index<n)
	{
		// generación de los vectores 
		vector_2[index]=(n-1)-index;
		vector_suma[index]=vectcor_1[index]+vector_2[index];	
	}

}

///////////////////////////////////
int main (int argc, char ** argv)
{	
	
	// declaraiones 
	
	int *hst_vector1, int *hst_vector2, *hst_resultado;
	
	int *dev_vector1, int *dev_vector2, *dev_resultado;
	int n=50;
	
	// reserva en el host
	hst_vector1=(int*)malloc(n*sizeof(int));
	hst_vector2=(int*)malloc(n*sizeof(int));
	hst_resultado=(int*)malloc(n*sizeof(int));
	
	// reserva en el device	
	hipMalloc((void**)&dev_vector1,n * sizeof(int));
	hipMalloc((void**)&dev_vector2,n * sizeof(int));
	hipMalloc((void**)&dev_resultado,n * sizeof(int));



	
	for(i=0; i<n; i++ )
	{
		hst_vector1[i]=i;
		hst_vector_2=0;
	}

	// envio de datos al device 
	hipMemcpy(dev_vector1,hst_vector1,n *sizeof(int), hipMemcpyHostToDevice);
	
	//lanzamiento del kernel 
	// calculo del numero de bloques

	int bloques = n/trheads;
	if (n%trheads)
	{	
	
		bloques=bloques+1;
	}

	printf("vector de %d elementos\n", n);

	printf("lanzamiento  con %d bloques de %d hilos(%d hilos)\n",bloques, trheads n);

	suma<<<bloques, trheads>>> (dev_vestor1,dev_vector2,dev_resultado,n);

	// captura de los datos del device

	hipMemcpy(hst_vector2,dev_vector2,n*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hst_resultado,dev_resultado,n*sizeof(int), hipMemcpyDeviceToHost);

	printf("vector1:\n")

	for(int i=0; i<n i++;)
	{
		
		printf("%2d",hst_vector1[i]);
	
	}
	
	printf("vector2:\n")

	for(int i=0; i<n i++;)
	{
		
		printf("%2d",hst_vector2[i]);
	
	}

	printf("vector resultado:\n")

	for(int i=0; i<n i++;)
	{
		
		printf("%2d",hst_resultado[i]);
	
	}

	printf("\n");
	// salida
	printf("***************************************************\n");
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;

}
