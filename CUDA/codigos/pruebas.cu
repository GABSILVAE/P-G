///////////////////////////////////////////////////////////////////////////
// PROGRAMACIÓN EN CUDA C/C++
// Curso Basico
// Agosto 2020
///////////////////////////////////////////////////////////////////////////
//
///usr/local/cuda/bin/nvcc Basico2.cu -o test
//./test
//
// includes
#include<stdio.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#define filas 3
#define columnas 3

__global__ void add(int *a, int *b, int *c) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int i = (columnas * y) + x;
	if(y<filas && x< columnas)
	{
		c[i] = a[i] + b[i];
	}
}	


int main() {

	int *hst_a, *hst_b, *hst_c;
	int *dev_a, *dev_b, *dev_c;

	hst_a=(int*)malloc(filas *columnas * sizeof(int));
	hst_b=(int*)malloc(filas *columnas * sizeof(int));
	hst_c=(int*)malloc(filas *columnas * sizeof(int));


	
	hipMalloc((void**)&dev_a, filas *columnas * sizeof(int));
	hipMalloc((void**)&dev_b, filas *columnas * sizeof(int));
	hipMalloc((void**)&dev_c, filas *columnas * sizeof(int));


	 for (int i = 0; i < filas; i++) {
		int cont = 0;
		for (int j = 0; j < columnas; j++) {
			hst_a[i * columnas+j]=1;
			hst_b[i * columnas+j]=2;
   			cont++;
  		}
 	}



	hipMemcpy(dev_a, hst_a, filas * columnas * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b, hst_b, filas * columnas * sizeof(int),hipMemcpyHostToDevice);



	dim3 grid(columnas, filas);

	add<<<grid, 2>>>(dev_a, dev_b, dev_c);

	hipMemcpy(hst_c, dev_c, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);

	for (int y = 0; y < filas; y++)
	{
		for (int x = 0; x < columnas; x++) {
			printf("%3d ", hst_c[x + y*columnas]);
		}
  		printf("\n");
 	}
	return 0;


}
