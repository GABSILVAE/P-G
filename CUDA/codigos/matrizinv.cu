///////////////////////////////////////////////////////////////////////////
// PROGRAMACIÓN EN CUDA C/C++
// Curso Basico
// Agosto 2020
///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// defines
#define COLUMNAS 20 // Numero de columnas -> eje x
#define FILAS 12     // Numero de filas    -> eje y


__global__ void nodiag_normalize(float *A, float *I, int n , int i)
{
	// KERNEL BIDIMENSIONAL: (X,Y)
	// indice de columna: EJE x
	int x =blockIdx.x * blockDim.x + threadIdx.x;
	// indice de fila: EJE y
	int y =  blockIdx.y * blockDim.y + threadIdx.y;	
	
	if (x<n && y<n){
		
	if ( x == i && x!=y){
			
		I[x*n + y ] /= A[i*n + i];
		A[x*n + y] /= A[i*n +i];
	}
	}
}




__global__ void diag_normalize(float *A, float *I, int n , int i)
{
	// KERNEL BIDIMENSIONAL: (X,Y)
	// indice de columna: EJE x
	int x =blockIdx.x * blockDim.x + threadIdx.x;
	// indice de fila: EJE y
	int y =  blockIdx.y * blockDim.y + threadIdx.y;	
	
	if (x<n && y<n){
		
	if ( x == y && x==i){
			
		I[x*n + y ] /= A[i*n + i];
		A[x*n + y] /= A[i*n + i];
	}

	}
}






__global__ void gaussjordan(float *A, float *I, int n , int i)
{
	// KERNEL BIDIMENSIONAL: (X,Y)
	// indice de columna: EJE x
	int x =blockIdx.x * blockDim.x + threadIdx.x;
	// indice de fila: EJE y
	int y =  blockIdx.y * blockDim.y + threadIdx.y;	
	
	if (x<n && y<n){
		
		if ( x != i ){

			I[x*n + y ] -= I[i*n + i] * A[x*n +i];

			if(y !=i){

				A[x*n + y] -= A[i*n + i] * A[x*n +i];
			}

		}

	}
}



__global__ void set_zero(float *A, float *I, int n , int i)
{
	// KERNEL BIDIMENSIONAL: (X,Y)
	// indice de columna: EJE x
	int x =blockIdx.x * blockDim.x + threadIdx.x;
	// indice de fila: EJE y
	int y =  blockIdx.y * blockDim.y + threadIdx.y;	
	
	if (x<n && y<n){
		
		if ( x != i ){

			I[x*n + y ] -= I[i*n + i] * A[x*n +i];

			if(y ==i){

				A[x*n + y] = 0;
			}

		}

	}
}


///////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
	// declaraciones
	float *hst_A, *hst_I;
	float *dev_A, *dev_I;
	
	int n =3;

	// reserva en el host
	hst_A = (float*)malloc(n*n * sizeof(float));
	hst_I = (float*)malloc(n*n * sizeof(float));
	
	//float d[n][n]={{1.0,2.0,3.0},{4.0,5.0,6.0},{7.0,8.0,9.0}};

	// reserva en el device
	hipMalloc((void**)&dev_A,  n*n* sizeof(float));
	hipMalloc((void**)&dev_I, n*n * sizeof(float));

	// incializacion
	for (int i = 1; i<9; i++)
	{
		hst_A[i] = i+1; // numeros consecutivos comenzando desde el 1
		hst_I[i] = 0;
	}

	// dimensiones del kernel
	// 1 Bloque
	dim3 Nbloques(1,1);
	
	// bloque bidimensional (x,y)
	// Eje x-> COLUMNAS
	// Eje y-> FILAS
	dim3 hilosB(3,3);

	// copia de datos hacia el device
	hipMemcpy(dev_A, hst_A, n*n * sizeof(float), hipMemcpyHostToDevice);

	for(int i=0; i<n; i++){

		nodiag_normalize <<<Nbloques, hilosB >>>(dev_A, dev_I,n ,i);
		diag_normalize <<<Nbloques, hilosB >>>(dev_A, dev_I,n ,i);
		gaussjordan <<<Nbloques, hilosB >>>(dev_A, dev_I,n ,i);
		set_zero <<<Nbloques, hilosB >>>(dev_A, dev_I,n ,i);



	}

	// llamada al kernel
	

	// recogida de datos desde el device
	hipMemcpy(hst_I, dev_I, n * sizeof(float), hipMemcpyDeviceToHost);

	// impresion de resultados
	printf("> MATRIZ ORIGINAL:\n");
	for (int i = 0; i<n; i++)
	{
		for (int j = 0; j<n; j++)
		{
			printf("%f ", hst_A[j + i*n]);
		}
		printf("\n");
	}
	printf("\n");
	printf("> MATRIZ FINAL:\n");
	for (int i = 0; i<n; i++)
	{
		for (int j = 0; j<n; j++)
		{
			printf("%f ", hst_I[j + i*n]);
		}
		printf("\n");
	}

	// salida del programa
	printf("\n<pulsa [INTRO] para finalizar>\n");
	getchar();
	return 0;
}
