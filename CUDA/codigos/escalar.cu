///////////////////////////////////////////////////////////////////////////
// PROGRAMACIÓN EN CUDA C/C++
// Curso Basico
// Agosto 2020
///////////////////////////////////////////////////////////////////////////
//
///usr/local/cuda/bin/nvcc Basico2.cu -o test
//./test
//
// includes
#include<stdio.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#define filas 3
#define columnas 3

__global__ void mult(int *a, int *num, int *b){

	int idx= blockIdx.x * blockDim.x + threadIdx.x;
	int idy= blockIdx.y * blockDim.y + threadIdx.y;
	int i = (columnas * idy) + idx;

	if(idy<filas && idx< columnas){

		b[i]=num[0] * a[i];	
	}

	


}


int main (int argc, char** argv)
{

	int *hst_a, *hst_num, *hst_b;

	int *dev_a, *dev_b, *dev_num;

	hst_a=(int*)malloc(filas*columnas *sizeof(int));
	hst_num=(int*)malloc(1 *sizeof(int)); 
	hst_b=(int*)malloc(filas*columnas *sizeof(int));



	hipMalloc((void**)&dev_a , filas*columnas * sizeof(int));
	hipMalloc((void**)&dev_num , 1 * sizeof(int));
	hipMalloc((void**)&dev_b , filas*columnas * sizeof(int));

	hst_num[0]=5;

	for (int i = 0; i < filas; i++) {
		
		for (int j = 0; j < columnas; j++) {
			hst_a[i * columnas+j]=1;
			hst_b[i * columnas+j]=0;
   			
  		}
 	}


	hipMemcpy(dev_a, hst_a, filas * columnas * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_num, hst_num, 1 * sizeof(int),hipMemcpyHostToDevice);

	dim3 grid(columnas, filas);

	mult<<<grid, 1>>>(dev_a, dev_num, dev_b);


	hipMemcpy(hst_b, dev_b, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);

	
	for (int y = 0; y < filas; y++)
	{
		for (int x = 0; x < columnas; x++) {
			printf("%3d ", hst_a[x + y*columnas]);
		}
  		printf("\n");
 	}


	for (int y = 0; y < filas; y++)
	{
		for (int x = 0; x < columnas; x++) {
			printf("%3d ", hst_b[x + y*columnas]);
		}
  		printf("\n");
 	}


	// salida del programa
	printf("\n<pulsa [INTRO] para finalizar>\n");
	getchar();
	return 0;










}
