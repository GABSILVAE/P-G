///////////////////////////////////////////////////////////////////////////
// PROGRAMACIÓN EN CUDA C/C++
// Curso Basico
// Agosto 2020
///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// defines
#define COLUMNAS 3 // Numero de columnas -> eje x
#define FILAS 3     // Numero de filas    -> eje y

__global__ void MatFinal(int *a, int *b, int *c)
{
	// KERNEL BIDIMENSIONAL: (X,Y)
	// indice de columna: EJE x
	int columna = threadIdx.x;
	// indice de fila: EJE y
	int fila = threadIdx.y;
	// KERNEL DE UN SOLO BLOQUE:
	// indice lineal
	int globalID = columna + fila * COLUMNAS;
	
	// Escritura en la matriz final
	c[globalID] = a[globalID]+b[globalID];
}
///////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
	// declaraciones
	int *hst_a, *hst_b, *hst_c;
	int *dev_a, *dev_b, *dev_c;

	// reserva en el host
	hst_a = (int*)malloc(FILAS*COLUMNAS * sizeof(int));
	hst_b = (int*)malloc(FILAS*COLUMNAS * sizeof(int));
	hst_c = (int*)malloc(FILAS*COLUMNAS * sizeof(int));

	// reserva en el device
	hipMalloc((void**)&dev_a, FILAS*COLUMNAS * sizeof(int));
	hipMalloc((void**)&dev_b, FILAS*COLUMNAS * sizeof(int));
	hipMalloc((void**)&dev_c, FILAS*COLUMNAS * sizeof(int));

	// incializacion
	for (int i = 0; i<FILAS*COLUMNAS; i++)
	{
		hst_a[i] = 1; // numeros consecutivos comenzando desde el 1
		hst_b[i] = 2;
	}

	// dimensiones del kernel
	// 1 Bloque
	dim3 Nbloques(1);
	
	// bloque bidimensional (x,y)
	// Eje x-> COLUMNAS
	// Eje y-> FILAS
	dim3 hilosB(COLUMNAS, FILAS);

	// copia de datos hacia el device
	hipMemcpy(dev_a, hst_a, FILAS*COLUMNAS * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, hst_b, FILAS*COLUMNAS * sizeof(int), hipMemcpyHostToDevice);

	// Numero de hilos
	printf("> KERNEL de 1 BLOQUE con %d HILOS:\n", COLUMNAS*FILAS);
	printf("  eje x -> %2d hilos\n  eje y -> %2d hilos\n", COLUMNAS, FILAS);

	// llamada al kernel
	MatFinal <<<Nbloques, hilosB >>>(dev_a, dev_b, dev_c);

	// recogida de datos desde el device
	hipMemcpy(hst_c, dev_c, FILAS*COLUMNAS * sizeof(int), hipMemcpyDeviceToHost);

	// impresion de resultados
	printf("> Matriz a:\n");
	for (int i = 0; i<FILAS; i++)
	{
		for (int j = 0; j<COLUMNAS; j++)
		{
			printf("%3d ", hst_a[j + i*COLUMNAS]);
		}
		printf("\n");
	}

	printf("\n");
	printf("> Matriz b:\n");
	for (int i = 0; i<COLUMNAS; i++)
	{
		for (int j = 0; j<FILAS; j++)
		{
			printf("%3d ", hst_b[j + i*COLUMNAS]);
		}
		printf("\n");
	} 

	printf("\n");
	printf("> Matriz c:\n");
	for (int i = 0; i<COLUMNAS; i++)
	{
		for (int j = 0; j<FILAS; j++)
		{
			printf("%3d ", hst_c[j + i*COLUMNAS]);
		}
		printf("\n");
	}





	// salida del programa
	printf("\n<pulsa [INTRO] para finalizar>\n");
	getchar();
	return 0;
}
