#include <stdio.h>
#include <hip/hip_runtime.h>
#define columnas 3
#define filas 3
__global__ void add(int *a, int *b, int *c) {

 int x = blockIdx.x * blockDim.x + threadIdx.x;
 int y = blockIdx.y * blockDim.y + threadIdx.y;
 int i = (columnas * y) + x;

 c[i] = a[i] + b[i];
}

int main() {
 int cont = 0;
 int i, j;
 // matrices en host
 int a[filas][columnas], b[filas][columnas], c[filas][columnas];

 // matrices en GPGPU
 int *dev_a, *dev_b, *dev_c;

 hipMalloc((void **) &dev_a, filas * columnas * sizeof(int));
 hipMalloc((void **) &dev_b, filas * columnas * sizeof(int));
 hipMalloc((void **) &dev_c, filas * columnas * sizeof(int));

 /* inicializando variables con datos foo*/
 for (i = 0; i < filas; i++) {
  cont = 0;
  for (j = 0; j < columnas; j++) {
   a[i][j] = 2;
   b[i][j] = 5;
   cont++;
  }
 }
 hipMemcpy(dev_a, a, filas * columnas * sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, filas * columnas * sizeof(int),hipMemcpyHostToDevice);

 // definiendo grid
 dim3 grid(columnas, filas);

 // grid del tamaño de la matriz, con un thread por bloque
 add<<<grid, 2>>>(dev_a, dev_b, dev_c);

 hipMemcpy(c, dev_c, filas * columnas * sizeof(int), hipMemcpyDeviceToHost);

 // imprimiendo
 for (int y = 0; y < filas; y++)
   {
  for (int x = 0; x < columnas; x++) {
   printf("[%d][%d]=%d ", y, x, c[y][x]);
  }
  printf("\n");
 }
 return 0;
}
