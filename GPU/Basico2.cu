﻿///////////////////////////////////////////////////////////////////////////
// PROGRAMACIÓN EN CUDA C/C++
// Curso Basico
// Agosto 2020
///////////////////////////////////////////////////////////////////////////
//
///usr/local/cuda/bin/nvcc Basico2.cu -o test
//./test
//
// includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void suma(int *vector_1, int *vector_2, int *vector_suma, int n)
{
	// identificador de hilo
	int myID = threadIdx.x;
	// generamos el vector 2
	vector_2[myID] = (n - 1) - myID;
	// escritura de resultados
	vector_suma[myID] = vector_1[myID] + vector_2[myID];
}

int main(int argc, char** argv)
{
	// declaraciones
	int *hst_vector1, *hst_vector2, *hst_resultado;
	int *dev_vector1, *dev_vector2, *dev_resultado;
	int n = 8;
	
	// reserva en el host
	hst_vector1 = (int*)malloc(n * sizeof(int));
	hst_vector2 = (int*)malloc(n * sizeof(int));
	hst_resultado = (int*)malloc(n * sizeof(int));

	// reserva en el device
	hipMalloc((void**)&dev_vector1, n * sizeof(int));
	hipMalloc((void**)&dev_vector2, n * sizeof(int));
	hipMalloc((void**)&dev_resultado, n * sizeof(int));

	// inicializacion de vectores
	for (int i = 0; i < n; i++)
	{
		hst_vector1[i] = i;
		hst_vector2[i] = 0;
	}

	// copia de datos hacia el device
	hipMemcpy(dev_vector1, hst_vector1, n * sizeof(int), hipMemcpyHostToDevice);

	// LANZAMIENTO DEL KERNEL
	suma <<< 1, n >>>(dev_vector1, dev_vector2, dev_resultado, n);

	// recogida de datos desde el device
	hipMemcpy(hst_vector2, dev_vector2, n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hst_resultado, dev_resultado, n * sizeof(int), hipMemcpyDeviceToHost);

	// impresion de resultados
	printf("VECTOR 1:\n");
	for (int i = 0; i < n; i++)
	{
		printf("%2d ", hst_vector1[i]);
	}
	printf("\n");
	printf("VECTOR 2:\n");
	for (int i = 0; i < n; i++)
	{
		printf("%2d ", hst_vector2[i]);
	}
	printf("\n");
	printf("SUMA:\n");
	for (int i = 0; i < n; i++)
	{
		printf("%2d ", hst_resultado[i]);
	}
	printf("\n");
	// salida
	printf("***************************************************\n");
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;
}