// Curso Basico
// Agosto 2020
///////////////////////////////////////////////////////////////////////////
//
///usr/local/cuda/bin/nvcc Basico2.cu -o test
//./test
//
// includes
#include<stdio.h>
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void prom(int *a, int *dato,int filas , int columnas, int n){

	int idy =  blockIdx.y * blockDim.y + threadIdx.y;
	int idx =  blockIdx.x * blockDim.x + threadIdx.x;
	//int i = (columnas * idy) + idx;
	
	if(idy<filas && idx <columnas){

		int sum=0;
		for(int j = 0; j<n ; j++){
			
			sum+= a[j];
		}
		dato[0]= sum/n;
	} 




}


int main (int argc, char** argv){


	int *hst_a, *hst_dato;
	int *dev_a, *dev_dato;

	int filas=6;
	int columnas=6;

	int n = filas*columnas;


	
	hst_a=(int*)malloc(n *sizeof(int));
	hst_dato=(int*)malloc(1 *sizeof(int));


	hipMalloc((void**)&dev_a, n * sizeof(int));
	hipMalloc((void**)&dev_dato , 1 * sizeof(int));



	for (int i = 0; i < filas; i++) {
		
		for (int j = 0; j < columnas; j++) {
			hst_a[i * columnas+j]=1;
			
  		}
 	}

	hipMemcpy(dev_a, hst_a, n * sizeof(int),hipMemcpyHostToDevice);
 
	dim3 grid(columnas, filas);

	prom<<<grid, 1>>>(dev_a, dev_dato,filas, columnas,n);

	hipMemcpy(hst_dato,dev_dato,1*sizeof(int), hipMemcpyDeviceToHost);



	for (int y = 0; y < filas; y++)
	{
		for (int x = 0; x < columnas; x++) {
			printf("%3d ", hst_a[x + y*columnas]);
		}
  		printf("\n");
 	}



	printf("vector resultado:\n");

	printf("%d",hst_dato[0]);



	
	// salida del programa
	printf("\n<pulsa [INTRO] para finalizar>\n");
	getchar();
	return 0;


}
