#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
// PROGRAMACIÓN EN CUDA C/C++
// Curso Basico
// Agosto 2020
///////////////////////////////////////////////////////////////////////////
//
///usr/local/cuda/bin/nvcc Basico2.cu -o test
//./test
//
// includes
#include<stdio.h>
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void suma(int *c, int *a, int *b, int filas, int columnas)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
 	int y = blockIdx.y * blockDim.y + threadIdx.y;
 	int i = (columnas * y) + x;

 	c[i] = a[i] + b[i];


}

int main (int argc, char** argv)
{
	int a[filas][columnas], b[filas][columnas], c[filas][columnas];
	int *dev_a, *dev_b, *dev_c;
	int filas=3;
	int columnas=3;
	int n=filas * columnas;





	hst_a=(int*)malloc(n * sizeof(int));
	hst_b=(int*)malloc(n * sizeof(int));
	hst_c=(int*)malloc(n * sizeof(int));

	hipMalloc((void**)&dev_a, n * sizeof(int));
	hipMalloc((void**)&dev_b, n * sizeof(int));
	hipMalloc((void**)&dev_c, n * sizeof(int));


	for (int i=0;i<filas;i++)
	{
		for(int j=0;j<columnas;j++)
		{
			hst_a[i * columnas+j]=1;
			hst_b[i * columnas+j]=2;		
		}
	}



	dim3 Nbloques(1);
	dim3 hilosb(columnas, filas);

	hipMemcpy(dev_a, hst_a, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, hst_b, n * sizeof(int), hipMemcpyHostToDevice);

	suma <<<Nbloques,hilosb>>> (dev_a,dev_b,dev_c,filas,columnas);
	
	hipMemcpy(hst_c, dev_c, n*sizeof(int), hipMemcpyDeviceToHost);

	
	printf("> Matriz a:\n");
	for (int i = 0; i<filas; i++)
	{
		for (int j = 0; j<columnas; j++)
		{
			printf("%3d ", hst_a[j + i*columnas]);
		}
		printf("\n");
	}


	


	printf("> Matriz b:\n");
	for (int i = 0; i<filas; i++)
	{
		for (int j = 0; j<columnas; j++)
		{
			printf("%3d ", hst_b[j + i*columnas]);
		}
		printf("\n");
	}


	printf("> Matriz c:\n");
	for (int i = 0; i<filas; i++)
	{
		for (int j = 0; j<columnas; j++)
		{
			printf("%3d ", hst_c[j + i*columnas]);
		}
		printf("\n");
	}





}
