#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define hilos 100

__global__ void dir (int *in,int *out,int n){
	
	int index= threadIdx.x +blockDim.x *blockIdx.x;
	

	if (index<n){
		
		out[0]=2 * in[0];
	}	
}


int main(int argc, char** argv)
{
	int *hst_in, *hst_out;
	int *dev_in, *dev_out;
	int n=10;

	hst_in=(int*)malloc(1*sizeof(int));
	hst_out=(int*)malloc(1*sizeof(int));
	


	hipMalloc((void**)&dev_in,1*sizeof(int));
	hipMalloc((void**)&dev_out,1*sizeof(int));


	hst_in[0]=2;

	hipMemcpy(dev_in,hst_in,1*sizeof(int), hipMemcpyHostToDevice);
 
	int bloques = n/hilos;
	
	if(n%hilos !=0){

		bloques=bloques+1;
	}


	printf("vector de %d elementos\n", n);

	printf("lanzamiento  con %d bloques de %d hilos(%d elementos)\n",bloques,hilos,n);

	dir<<<bloques, hilos>>> (dev_in,dev_out,n);

	hipMemcpy(hst_in,dev_in,1*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hst_out,dev_out,1*sizeof(int), hipMemcpyDeviceToHost);




	printf("in:\n");
	
	printf("%d \n",hst_in[0]);


	printf("out:\n");
	
	printf("%d \n",hst_out[0]);


	printf("\n");
	// salida
	printf("***************************************************\n");
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;





  	
} 
