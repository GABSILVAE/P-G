///////////////////////////////////////////////////////////////////////////
// PROGRAMACIÓN EN CUDA C/C++
// Curso Basico
// Agosto 2020
///////////////////////////////////////////////////////////////////////////
//
///usr/local/cuda/bin/nvcc Basico2.cu -o test
//./test
//
// includes
#include<stdio.h>
#include<stdio.h>
#include<hip/hip_runtime.h>

#define hilos 10 // numero de hilos por bloque 

__global__ void suma(int *vector_1, int *dato, int n){

	// kernel multibloque 
	// identificaador global de cada hilo en el bloque 
	int index =threadIdx.x + blockDim.x * blockIdx.x;
	vector_1[index] = index;
	//truncamiento de los hilos solo trabajan los hilos nesesarios 
	if(index<n)
	{
		int sum=0;
		for(int i=0;i<n;i++){
			sum += vector_1[i];
		}
		dato[0]=sum;
	}

}

///////////////////////////////////
int main (void)
{	
	
	// declaraiones 
	
	int *hst_vector1, *hst_dato;
	
	int *dev_vector1, *dev_dato;

	int n=5;
	
	// reserva en el host
	hst_vector1=(int*)malloc(n*sizeof(int));
	hst_dato=(int*)malloc(1*sizeof(int));
	
	
	// reserva en el device	
	hipMalloc((void**)&dev_vector1,n * sizeof(int));
	hipMalloc((void**)&dev_dato,1 * sizeof(int));
	


	
	for(int i=0; i<n; i++)
	{
		hst_vector1[i]=0;
		
	}

	// envio de datos al device 
	//cudaMemcpy(dev_vector1,hst_vector1,n *sizeof(int), cudaMemcpyHostToDevice);
	
	//lanzamiento del kernel 
	// calculo del numero de bloques

	int bloques = n/hilos;

	if (n%hilos !=0)
	{	
	
		bloques=bloques+1;
	}

	printf("vector de %d elementos\n", n);

	printf("lanzamiento  con %d bloques de %d hilos(%d elementos)\n",bloques,hilos,n);

	suma<<<bloques, hilos>>> (dev_vector1,dev_dato,n);

	// captura de los datos del device

	hipMemcpy(hst_vector1,dev_vector1,n*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hst_dato,dev_dato,1*sizeof(int), hipMemcpyDeviceToHost);

	/*printf("vector1:\n");
	
	
	
	for(int i=0; i<n; i++)
	{
		
		printf("%d",hst_vector1[i]);
	
	}
	*/
	
	printf("vector resultado:\n");

	printf("%d",hst_dato[0]);
	
	

	printf("\n");
	// salida
	printf("***************************************************\n");
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;

}
