///////////////////////////////////////////////////////////////////////////
// PROGRAMACIÓN EN CUDA C/C++
// Curso Basico
// Agosto 2020
///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// defines
#define COLUMNAS 3 // Numero de columnas -> eje x
#define FILAS 3     // Numero de filas    -> eje y

__global__ void MatFinal(int *a, int *num, int *c)
{
	// KERNEL BIDIMENSIONAL: (X,Y)
	// indice de columna: EJE x
	int columna = threadIdx.x;
	// indice de fila: EJE y
	int fila = threadIdx.y;
	// KERNEL DE UN SOLO BLOQUE:
	// indice lineal
	int globalID = columna + fila * COLUMNAS;
	
	// Escritura en la matriz final
	c[globalID] = num[0] *a [globalID];
}
///////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
	// declaraciones
	int *hst_a, *hst_c, *hst_num;
	int *dev_a, *dev_c, *dev_num;

	// reserva en el host
	hst_a = (int*)malloc(FILAS*COLUMNAS * sizeof(int));
	hst_num = (int*)malloc(1* sizeof(int));
	hst_c = (int*)malloc(FILAS*COLUMNAS * sizeof(int));

	// reserva en el device
	hipMalloc((void**)&dev_a, FILAS*COLUMNAS * sizeof(int));
	hipMalloc((void**)&dev_num, 1 * sizeof(int));
	hipMalloc((void**)&dev_c, FILAS*COLUMNAS * sizeof(int));

	hst_num[0]=3;

	// incializacion
	for (int i = 0; i<FILAS*COLUMNAS; i++)
	{
		hst_a[i] = 1; // numeros consecutivos comenzando desde el 1
		
	}

	// dimensiones del kernel
	// 1 Bloque
	dim3 Nbloques(1);
	
	// bloque bidimensional (x,y)
	// Eje x-> COLUMNAS
	// Eje y-> FILAS
	dim3 hilosB(COLUMNAS, FILAS);

	// copia de datos hacia el device
	hipMemcpy(dev_a, hst_a, FILAS*COLUMNAS * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_num, hst_num, 1 * sizeof(int), hipMemcpyHostToDevice);
	
	// Numero de hilos
	printf("> KERNEL de 1 BLOQUE con %d HILOS:\n", COLUMNAS*FILAS);
	printf("  eje x -> %2d hilos\n  eje y -> %2d hilos\n", COLUMNAS, FILAS);

	// llamada al kernel
	MatFinal <<<Nbloques, hilosB >>>(dev_a, dev_num, dev_c);

	// recogida de datos desde el device
	hipMemcpy(hst_c, dev_c, FILAS*COLUMNAS * sizeof(int), hipMemcpyDeviceToHost);

	// impresion de resultados
	printf("> Matriz a:\n");
	for (int i = 0; i<FILAS; i++)
	{
		for (int j = 0; j<COLUMNAS; j++)
		{
			printf("%3d ", hst_a[j + i*COLUMNAS]);
		}
		printf("\n");
	}

	 

	printf("\n");
	printf("> Matriz c:\n");
	for (int i = 0; i<COLUMNAS; i++)
	{
		for (int j = 0; j<FILAS; j++)
		{
			printf("%3d ", hst_c[j + i*COLUMNAS]);
		}
		printf("\n");
	}





	// salida del programa
	printf("\n<pulsa [INTRO] para finalizar>\n");
	getchar();
	return 0;
}
