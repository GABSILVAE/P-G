﻿///////////////////////////////////////////////////////////////////////////
// PROGRAMACIÓN EN CUDA C/C++
// Curso Basico
// Agosto 2020
///////////////////////////////////////////////////////////////////////////
//
///usr/local/cuda/bin/nvcc Basico2.cu -o test
//./test
//
// includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define hilos 10 // numero de hilos por bloque 
__global__ void suma(int *vector_1, int *vector_2, int *vector_suma, int n)
{
	// kernel multibloque 
	// identificaador global de cada hilo en el bloque 
	int index =threadIdx.x + blockDim.x * blockIdx.x;
	//truncamiento de los hilos solo trabajan los hilos nesesarios 
	if(index<n)
	{
		// generación de los vectores 
		vector_2[index]=(n-1)-index;
		vector_suma[index]=vector_1[index]+vector_2[index];	
	}

}

int main(int argc, char** argv)
{
	// declaraciones
	int *hst_vector1, *hst_vector2, *hst_resultado;
	int *dev_vector1, *dev_vector2, *dev_resultado;
	int n = 50;
	
	// reserva en el host
	hst_vector1 = (int*)malloc(n * sizeof(int));
	hst_vector2 = (int*)malloc(n * sizeof(int));
	hst_resultado = (int*)malloc(n * sizeof(int));

	// reserva en el device
	hipMalloc((void**)&dev_vector1, n * sizeof(int));
	hipMalloc((void**)&dev_vector2, n * sizeof(int));
	hipMalloc((void**)&dev_resultado, n * sizeof(int));

	// inicializacion de vectores
	for (int i = 0; i < n; i++)
	{
		hst_vector1[i] = i;
		hst_vector2[i] = 0;
	}
	


	// copia de datos hacia el device
	hipMemcpy(dev_vector1, hst_vector1, n * sizeof(int), hipMemcpyHostToDevice);

	int bloques = n/hilos;
	if (n%hilos)
	{	
	
		bloques=bloques+1;
	}

	printf("vector de %d elementos\n", n);

	printf("lanzamiento  con %d bloques de %d hilos(%d hilos)\n",bloques,hilos,n);

	// LANZAMIENTO DEL KERNEL
	suma <<< bloques, hilos >>>(dev_vector1, dev_vector2, dev_resultado, n);

	
	// recogida de datos desde el device
	hipMemcpy(hst_vector2, dev_vector2, n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hst_resultado, dev_resultado, n * sizeof(int), hipMemcpyDeviceToHost);

	// impresion de resultados
	printf("VECTOR 1:\n");
	for (int i = 0; i < n; i++)
	{
		printf("%2d ", hst_vector1[i]);
	}
	printf("\n");
	printf("VECTOR 2:\n");
	for (int i = 0; i < n; i++)
	{
		printf("%2d ", hst_vector2[i]);
	}
	printf("\n");
	printf("SUMA:\n");
	for (int i = 0; i < n; i++)
	{
		printf("%2d ", hst_resultado[i]);
	}
	printf("\n");
	// salida
	printf("***************************************************\n");
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;
}



