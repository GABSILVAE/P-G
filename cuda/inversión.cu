#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void inversion(int *vector_1, int *vector_2, int n)
{
	int index= threadIdx.x;
	vector_1[index]=index;
	vector_2[index]=vector_1[((n-1)-index)];
}

int main (int argc, char** argv){
	
	
	// declaraciones
	int *hst_vector_1, *hst_vector_2;
	int *dev_vector_1, *dev_vector_2;
	int n=8;
	
	// reserva de memoria  en el host
	hst_vector_1=(int*)malloc(n*sizeof(int));
	hst_vector_2=(int*)malloc(n*sizeof(int));
	
	// reserva de memoria en el device
	hipMalloc ((void**)&dev_vector_1, n*sizeof(int));
	hipMalloc ((void**)&dev_vector_2, n*sizeof(int));
	
	// inicialización de los vectore en el host
	for (int i=0; i<n; i++){
		
		hst_vector_1[i]=0;
		hst_vector_2[i]=0;
		
	}
	
	// copia de los datos
	
	inversion <<<1,n>>> (dev_vector_1, dev_vector_2, n);
	
	// envio de datos del device a host
	
	hipMemcpy(hst_vector_1,dev_vector_1, n*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hst_vector_2,dev_vector_2, n*sizeof(int), hipMemcpyDeviceToHost);}

	// imprimiendo los datos
	
	printf ("VECTOR1: \n");
	for(int i=0; i<0; i++){
		
		printf("%2d", hst_vector_1[i]);
	}
	
	printf("\n");
	
	printf("VECTOR2: \n")
	
	for(int i=0; i< n; i++){
		
		printf("2%d", hst_vector_2[i]);
		
		
	}
	
	return 0;
	
}
