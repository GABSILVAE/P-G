﻿///////////////////////////////////////////////////////////////////////////
// PROGRAMACIÓN EN CUDA C/C++
// Curso Basico
// Agosto 2020
///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// defines
#define BLOCK 10 // Lanzamiento con bloques de 'BLOCK' hilos

__global__ void suma(int *vector_1, int *vector_2, int *vector_suma, int n)
{
	// KERNEL 'MULTIBLOQUE'
	// identificador global de cada hilo
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	// solo trabajan n hilos
	// el resto no debe hacer nada
	if (myID < n)
	{
		// generamos el vector 2
		vector_2[myID] = (n - 1) - myID;
		
		// escritura de resultados
		vector_suma[myID] = vector_1[myID] + vector_2[myID];
	}
}
int main(int argc, char** argv)
{
	// declaraciones
	int *hst_vector1, *hst_vector2, *hst_resultado;
	int *dev_vector1, *dev_vector2, *dev_resultado;
	int n = 25;

	// reserva en el host
	hst_vector1 = (int*)malloc(n * sizeof(int));
	hst_vector2 = (int*)malloc(n * sizeof(int));
	hst_resultado = (int*)malloc(n * sizeof(int));

	// reserva en el device
	hipMalloc((void**)&dev_vector1, n * sizeof(int));
	hipMalloc((void**)&dev_vector2, n * sizeof(int));
	hipMalloc((void**)&dev_resultado, n * sizeof(int));

	// inicializacion de vectores
	for (int i = 0; i < n; i++)
	{
		hst_vector1[i] = i;
		hst_vector2[i] = 0;
	}

	// copia de datos hacia el device
	hipMemcpy(dev_vector1, hst_vector1, n * sizeof(int), hipMemcpyHostToDevice);

	// LANZAMIENTO DEL KERNEL
	// calculamos el numero de bloques
	int bloques = n / BLOCK;
	if (n%BLOCK != 0)
		// Si el tamaño del vector no es multiplo del tamaño del bloque
		// lanzamos un bloque completo adicional
	{
		bloques = bloques + 1;
	}

	printf("> Vector de %d elementos\n", n);
	printf("> Lanzamiento con %d bloques de %d hilos (%d hilos)\n", bloques, BLOCK, bloques*BLOCK);
	
	suma <<< bloques, BLOCK >>>(dev_vector1, dev_vector2, dev_resultado, n);

	// recogida de datos desde el device
	hipMemcpy(hst_vector2, dev_vector2, n * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hst_resultado, dev_resultado, n * sizeof(int), hipMemcpyDeviceToHost);

	// impresion de resultados
	printf("VECTOR 1:\n");
	for (int i = 0; i < n; i++)
	{
		printf("%2d ", hst_vector1[i]);
	}
	printf("\n");
	printf("VECTOR 2:\n");
	for (int i = 0; i < n; i++)
	{
		printf("%2d ", hst_vector2[i]);
	}
	printf("\n");
	printf("SUMA:\n");
	for (int i = 0; i < n; i++)
	{
		printf("%2d ", hst_resultado[i]);
	}
	printf("\n");
	// salida
	printf("\n<pulsa [INTRO] para finalizar>\n");
	getchar();
	return 0;
}
///////////////////////////////////////////////////////////////////////////
// HOST: funcion llamada desde el host y ejecutada en el host
__host__ void propiedades_Device(int deviceID)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, deviceID);
	// calculo del numero de cores (SP)
	int cudaCores = 0;
	int SM = deviceProp.multiProcessorCount;
	int major = deviceProp.major;
	int minor = deviceProp.minor;
	switch (major)
	{
	case 1:
		//TESLA
		cudaCores = 8;
		break;
	case 2:
		//FERMI
		if (minor == 0)
			cudaCores = 32;
		else
			cudaCores = 48;
		break;
	case 3:
		//KEPLER
		cudaCores = 192;
		break;
	case 5:
		//MAXWELL
		cudaCores = 128;
		break;
	case 6:
		//PASCAL
		cudaCores = 64;
		break;
	case 7:
		//VOLTA (7.0) TURING (7.5)
		cudaCores = 64;
		break;
	case 8:
		//AMPERE
		cudaCores = 64;
		break;
	default:
		//ARQUITECTURA DESCONOCIDA
		cudaCores = 0;
		printf("!!!!!dispositivo desconocido!!!!!\n");
	}
	// presentacion de propiedades
	printf("***************************************************\n");
	printf("DEVICE %d: %s\n", deviceID, deviceProp.name);
	printf("***************************************************\n");
	printf("> Capacidad de Computo            \t: %d.%d\n", major, minor);
	printf("> No. de MultiProcesadores        \t: %d \n", SM);
	printf("> No. de CUDA Cores (%dx%d)       \t: %d \n", cudaCores, SM, cudaCores*SM);
	printf("> No. maximo de Hilos (por bloque)\t: %d\n", deviceProp.maxThreadsPerBlock);
	printf(" [eje x -> %d]\n [eje y -> %d]\n [eje z -> %d]\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("> No. maximo de Bloques (por eje):\n");
	printf(" [eje x -> %d]\n [eje y -> %d]\n [eje z -> %d]\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
	printf("***************************************************\n");
}
