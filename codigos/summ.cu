///////////////////////////////////////////////////////////////////////////
// PROGRAMACIÓN EN CUDA C/C++
// Curso Basico
// Agosto 2020
///////////////////////////////////////////////////////////////////////////
//
///usr/local/cuda/bin/nvcc Basico2.cu -o test
//./test
//
// includes
#include<stdio.h>
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void suma(int *c, int *a, int *b, int filas, int columnas)
{
	int idx=threadIdx.x + blockDim.x * blockIdx.x;
	int idy=threadIdx.y + blockDim.y * blockIdx.y;

	int index =idy*columnas+idx;

	if(idy<filas && idx<columnas){
	
		c[index]=a[index]+b[index];	

	}

}

int main (int argc, char** argv)
{
	int *hst_a, *hst_b, *hst_c;
	int *dev_a, *dev_b, *dev_c;
	int filas=3;
	int columnas=3;
	int n=filas * columnas;

	hst_a=(int*)malloc(n * sizeof(int));
	hst_b=(int*)malloc(n * sizeof(int));
	hst_c=(int*)malloc(n * sizeof(int));

	hipMalloc((void**)&dev_a, n * sizeof(int));
	hipMalloc((void**)&dev_b, n * sizeof(int));
	hipMalloc((void**)&dev_c, n * sizeof(int));

	
	for (int i=0;i<filas;i++)
	{
		for(int j=0;j<columnas;j++)
		{
			hst_a[i * columnas+j]=1;
			hst_b[i * columnas+j]=2;		
		}
	}


	dim3 Nbloques(1);
	dim3 hilosb(columnas, filas);

	hipMemcpy(dev_a, hst_a, n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, hst_b, n * sizeof(int), hipMemcpyHostToDevice);

	suma <<<Nbloques,hilosb>>> (dev_a,dev_b,dev_c,filas,columnas);
	
	

	hipMemcpy(hst_c, dev_c, n*sizeof(int), hipMemcpyDeviceToHost);

	/*printf("matiz a:\n");

	for (int i=0;i<filas;i++)
	{
		for(int j=0;j<columnas;j++)
		{
			printf("%d \n"hst_a[i][j]);
					
		}
	}

	printf("matiz b:\n");

	for (int i=0;i<filas;i++)
	{
		for(int j=0;j<columnas;j++)
		{
			printf("%d \n"hst_b[i][j]);
					
		}
	}


	printf("matiz c:\n");

	for (int i=0;i<filas;i++)
	{
		for(int j=0;j<columnas;j++)
		{
			printf("%d \n"hst_c[i][j]);
					
		}
	}

*/
	printf("\n");
	// salida
	printf("***************************************************\n");
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;



}












