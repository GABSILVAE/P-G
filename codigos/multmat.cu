///////////////////////////////////////////////////////////////////////////
// PROGRAMACIÓN EN CUDA C/C++
// Curso Basico
// Agosto 2020
///////////////////////////////////////////////////////////////////////////
//
///usr/local/cuda/bin/nvcc Basico2.cu -o test
//./test
//
// includes
#include<stdio.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#define hilos 9

__global__ void mult(int *a, int *b, int *c, int columnas, int filas )
{

	int idy =  blockIdx.y * blockDim.y + threadIdx.y;
	int idx =  blockIdx.x * blockDim.x + threadIdx.x;
	int i =  idy * columnas + idx;
	
	if(idy<filas && idx <columnas){

		int sum=0;
		for(int j = 0; j<columnas ; j++){
			
			sum+= a[idy * columnas +j] * b[j * columnas + idx];
		}
		c[i]= sum;
	} 

	

}


int main(int argc, char ** argv)
{


	int *hst_a, *hst_b, *hst_c;
	
	int *dev_a, *dev_b, *dev_c;
	
	int filas=640;
	int columnas=480;

	int n = filas*columnas; 
	
	hst_a=(int*)malloc(n *sizeof(int));
	hst_b=(int*)malloc(n *sizeof(int)); 
	hst_c=(int*)malloc(n *sizeof(int));


	hipMalloc((void**)&dev_a, n * sizeof(int));
	hipMalloc((void**)&dev_b, n * sizeof(int));
	hipMalloc((void**)&dev_c, n * sizeof(int));


	for (int i = 0; i < filas; i++) {
		
		for (int j = 0; j < columnas; j++) {
			hst_a[i * columnas+j]=1;
			hst_b[i * columnas+j]=2;
   			
  		}
 	}

	hipMemcpy(dev_a, hst_a,n * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b, hst_b,n * sizeof(int),hipMemcpyHostToDevice);


	
	dim3 grid(columnas, filas);

	

	printf("matrices  de %d elementos\n", n);

	

	mult<<<grid, 6>>> (dev_a,dev_b,dev_c,filas,columnas);


	hipMemcpy(hst_c, dev_c, n * sizeof(int), hipMemcpyDeviceToHost);

	for (int y = 0; y < filas; y++)
	{
		for (int x = 0; x < columnas; x++) {
			printf("%3d ", hst_a[x + y*columnas]);
		}
  		printf("\n");
 	}


	for (int y = 0; y < filas; y++)
	{
		for (int x = 0; x < columnas; x++) {
			printf("%3d ", hst_b[x + y*columnas]);
		}
  		printf("\n");
 	}


	for (int y = 0; y < filas; y++)
	{
		for (int x = 0; x < columnas; x++) {
			printf("%3d ", hst_c[x + y*columnas]);
		}
  		printf("\n");
 	}


	// salida del programa
	printf("\n<pulsa [INTRO] para finalizar>\n");
	getchar();
	return 0;



}
